#include "hip/hip_runtime.h"
//general parts
#include <stdio.h>
#include <vector>
#include <memory>
#include <string.h>
#include <chrono>
#include <thread>
#include <iostream>
#ifndef __STDC_FORMAT_MACROS
#define __STDC_FORMAT_MACROS
#endif
#include <inttypes.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include "utils_VkFFT.h"
int kernelCount = 0;

VkFFTResult run_FFT_benchmark(VkGPU* vkGPU, uint64_t num, int num_execs = 1, int useLUT = 1)
{
	VkFFTResult resFFT = VKFFT_SUCCESS;
	hipError_t res = hipSuccess;
	//printf("0 - VkFFT FFT + iFFT C2C benchmark 1D batched in single precision. Size: %lu, using LUT? %d\n", num, useLUT);
	//double benchmark_result = 0;//averaged result = sum(system_size/iteration_time)/num_benchmark_samples
	//memory allocated on the CPU once, makes benchmark completion faster + avoids performance issues connected to frequent allocation/deallocation.
	float* buffer_input = (float*)malloc((uint64_t)4 * 2 * (uint64_t)pow(2, 27));
	
	if (!buffer_input) 
		return VKFFT_ERROR_MALLOC_FAILED;
	for (uint64_t i = 0; i < 2 * (uint64_t)pow(2, 27); i++) {
		buffer_input[i] = (float)(2 * ((float)rand()) / RAND_MAX - 1.0);
	}
		
	const int num_runs = 1;
	double run_time[num_runs];
	for (uint64_t r = 0; r < num_runs; r++) {
		//Configuration + FFT application .
		VkFFTConfiguration *configuration = (VkFFTConfiguration *)malloc(num_execs * sizeof(VkFFTConfiguration));
		VkFFTApplication *app = (VkFFTApplication *)malloc(num_execs * sizeof(VkFFTApplication));
		for(int i = 0; i < num_execs; ++i) {
			configuration[i] = {};
			app[i] = {};
			//FFT + iFFT sample code.
			//Setting up FFT configuration for forward and inverse FFT.
			configuration[i].FFTdim = 1; //FFT dimension, 1D, 2D or 3D (default 1).
			configuration[i].size[0] = num;
			configuration[i].numberBatches = 1;
			
			configuration[i].device = &vkGPU->device;
			//Allocate buffer for the input data.
			uint64_t bufferSize = (uint64_t)sizeof(float) * 2 * configuration[i].size[0] * configuration[i].numberBatches;
			hipFloatComplex* buffer = 0;
			res = hipMalloc((void**)&buffer, bufferSize);
			if (res != hipSuccess) 
				return VKFFT_ERROR_FAILED_TO_ALLOCATE;
			configuration[i].buffer = (void**)&buffer;

			configuration[i].bufferSize = &bufferSize;

			//Sample buffer transfer tool. Uses staging buffer of the same size as destination buffer, which can be reduced if transfer is done sequentially in small buffers.
			res = hipMemcpy(buffer, buffer_input, bufferSize, hipMemcpyHostToDevice);
			if (res != hipSuccess) 
				return VKFFT_ERROR_FAILED_TO_COPY;
			
			configuration[i].num_streams = 1;
			hipStream_t *stream = (hipStream_t *)malloc(sizeof(hipStream_t));
			hipStreamCreate(stream);
			configuration[i].stream = stream;
			configuration[i].useLUT = useLUT;
			printf("Created stream\n");
			//Initialize applications. This function loads shaders, creates pipeline and configures FFT based on configuration file. No buffer allocations inside VkFFT library.  
			resFFT = initializeVkFFT(&app[i], configuration[i]);
			fflush(stdout);
			if (resFFT != VKFFT_SUCCESS) 
				return resFFT;
		}
		printf("Setup %d configs\n", num_execs);

		//Submit FFT+iFFT.
		uint64_t num_iter = 1;
		double totTime = 0;

		VkFFTLaunchParams *launchParams = (VkFFTLaunchParams*)malloc(sizeof(VkFFTLaunchParams));
		//resFFT = performVulkanFFTiFFT(vkGPU, &app, &launchParams, num_iter, &totTime);
		hipError_t res = hipSuccess;
		std::chrono::steady_clock::time_point timeSubmit = std::chrono::steady_clock::now();
		for(int j = 0; j < num_execs; ++j) {
			launchParams[j] = {};
			for (uint64_t i = 0; i < num_iter; i++) {
				resFFT = VkFFTAppend(&app[j], -1, &launchParams[j]);
				if (resFFT != VKFFT_SUCCESS) return resFFT;
				resFFT = VkFFTAppend(&app[j], 1, &launchParams[j]);
				if (resFFT != VKFFT_SUCCESS) return resFFT;
			}
		}
		res = hipDeviceSynchronize();
		if (res != hipSuccess) return VKFFT_ERROR_FAILED_TO_SYNCHRONIZE;
		std::chrono::steady_clock::time_point timeEnd = std::chrono::steady_clock::now();
		totTime = std::chrono::duration_cast<std::chrono::microseconds>(timeEnd - timeSubmit).count() * 0.001;
		totTime = totTime / num_iter;
		
		run_time[r] = totTime;
		if (r == num_runs - 1) {
			double std_error = 0;
			double avg_time = 0;
			for (uint64_t t = 0; t < num_runs; t++) {
				avg_time += run_time[t];
			}
			avg_time /= num_runs;
			for (uint64_t t = 0; t < num_runs; t++) {
				std_error += (run_time[t] - avg_time) * (run_time[t] - avg_time);
			}
			std_error = sqrt(std_error / num_runs);
			/*uint64_t num_tot_transfers = 0;
			for (uint64_t i = 0; i < configuration.FFTdim; i++)
				num_tot_transfers += app.localFFTPlan->numAxisUploads[i];
			num_tot_transfers *= 4;*/

			printf("VkFFT - Size: %" PRIu64 ", Batches: %" PRIu64 ", avg_time_per_step: %0.3f ms, num_iter: %" PRIu64 ", num streams: %d, Using LUT? %d\n", configuration[0].size[0], configuration[0].numberBatches, avg_time, num_iter, num_execs, useLUT);
			//benchmark_result += ((double)bufferSize / 1024) / avg_time;
		}
/*
		for(int j = 0; j < num_execs; ++j) {
			hipFree(buffer[j]);
			deleteVkFFT(&app[j]);
		}
*/
	}
	free(buffer_input);
	//benchmark_result /= 25;
	//printf("Benchmark score VkFFT: %" PRIu64 "\n", (uint64_t)(benchmark_result));
	return resFFT;
}

int main(int argc, char *argv[]) {
	uint64_t num = 65536 / 4;
	int streams = 2;
	int useLUT = 1;
	if(argc >= 2)
		num = atoi(argv[1]);
	if(argc >= 3)
		streams = atoi(argv[2]);
	if(argc >= 4)
		useLUT = atoi(argv[3]);
	VkGPU vkGPU;
	hipCtxGetCurrent ( &vkGPU.context );
	hipCtxGetDevice ( &vkGPU.device );
	VkFFTResult res = run_FFT_benchmark(&vkGPU, num, streams, useLUT);
	printf("Result: %d\n", res);
	return 0;
}
